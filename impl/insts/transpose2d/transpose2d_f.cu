#include "hip/hip_runtime.h"
#include "transpose2d.cuh"

static __global__ void kerd__transpose2d__simple(
	uint x0_t, uint X0, float * x0,
	//
	uint    Y,
	float * y,
	//
	uint * ts__d, uint mega_t,
	//
	uint Ax, uint Ay, uint C0)
{
	uint thx = threadIdx.x + blockIdx.x * blockDim.x;
	uint thy = threadIdx.y + blockIdx.y * blockDim.y;

	//	thx = Ax*Ay
	uint _ax = thx % Ax;
	uint _ay = (thx-_ax)/Ax;

	//	thy = C0*GRAND_T
	uint _c0 = thx % C0;
	uint __t = (thx-_c0)/C0;

	if (_ay < Ay && _ax < Ax && _c0 < C0 && __t < GRAND_T) {
		uint tx0 = t_MODE(__t, mega_t-x0_t);
		uint ty  = t_MODE(__t, mega_t     );
		//
		uint A  = ty*X0 + _c0*(Ax*Ay) + _ay*Ax + _ax;
		uint At = ty*X0 + _c0*(Ax*Ay) + _ax*Ay + _ay;
		//
		y[At] = x0[A];
	}
};

//	---------------------------------------------------------------------------------

void transpose2d__f(Inst_t * inst, float ** x__d, uint * ts__d, uint mega_t) {
	uint * params = inst->params;
	uint \
		Ax =params[0],	\
		Ay =params[1],	\
		C0 =params[2];
	//
	uint x0_t = inst->x_t[0];
	//
	bool x0_existe = (mega_t != 0 ? true : (x0_t != 1));
	//
	ASSERT(x0_existe);
	//
	if (x0_existe) {
		kerd__transpose2d__simple<<<dim3(KERD((Ax*Ay),16), KERD((C0*GRAND_T),16)), dim3(16,16)>>>(
			inst->x_t[0], inst->x_Y[0], x__d[0],
			//
			inst->Y,
			inst->y__d,
			//
			ts__d, mega_t,
			//
			Ax, Ay, C0
		);
	} else {
		inst_zero_mega_t(inst, mega_t);
	}
};