#include "hip/hip_runtime.h"
#include "mdl.cuh"

#include "../impl_template/tmpl_etc.cu"

__global__
static void kerd_p1e5(float * p__d, uint p, float _1E5) {
	p__d[p] += _1E5;
};

static void plus_1e5(float * p__d, uint p, float _1E5) {
	kerd_p1e5<<<1,1>>>(p__d, p, _1E5);
	ATTENDRE_CUDA();
};

//	---------------------------------------------------

__global__
static void kerd_lire(float * p__d, uint p, float * val) {
	val[0] = p__d[p];
};

static float lire(float * p__d, uint p) {
	float * val = cudalloc<float>(1);
	kerd_lire<<<1,1>>>(p__d, p, val);
	ATTENDRE_CUDA();
	//
	float * _ret = gpu_vers_cpu<float>(val, 1);
	float ret = _ret[0];
	free(_ret);cudafree<float>(val);
	//
	return ret;
};

void tester_le_model(Mdl_t * mdl, BTCUSDT_t * btcusdt) {
	uint ts[GRAND_T];
	FOR(0, t, GRAND_T) ts[t] = rand() % (btcusdt->T - MEGA_T);
	uint * ts__d = cpu_vers_gpu<uint>(ts, GRAND_T);
	//
	mdl_verif(mdl, btcusdt);
	//
	//	mdl_plume_poid(mdl);
	//
	float * grad_1e5[mdl->insts];
	FOR(0, i, mdl->insts) grad_1e5[i] = alloc<float>(mdl->inst[i]->P);
	//
	//
	mdl_allez_retour(mdl, btcusdt, ts__d);
	//
	//
	INIT_CHRONO(s)
	DEPART_CHRONO(s)
	//
	uint testés = 1;
	//
	float S = mdl_S(mdl, btcusdt, ts__d);
	float _1E5 = 5e-3;
	uint lp = 0;
	FOR(0, i, mdl->insts) {
		printf("#### INSTRUCTION %i (%s Y=%i) ####\n",
			i, 
			inst_Nom[mdl->inst[i]->ID], mdl->inst[i]->Y
		);
		FOR(0, p, mdl->inst[i]->P) {
			plus_1e5(mdl->inst[i]->p__d, p, _1E5);
			float S1e5 = mdl_S(mdl, btcusdt, ts__d);
			plus_1e5(mdl->inst[i]->p__d, p, -_1E5);

			//
			grad_1e5[i][p] = (S1e5 - S)/_1E5;
			//printf("%i|%i| %f --- %f (%f)\n", );

			float a=grad_1e5[i][p], b=lire(mdl->inst[i]->dp__d, p);
			printf("%i| ", p);
			PLUME_CMP(a, b);
			if (b != 0) printf(" (x%+f) ", a/b);
			printf(" (%+fs)", (float)testés / VALEUR_CHRONO(s));
			printf(" |%i  inst=%i|\n", lp++, i);
			testés++;
		};
	};
	printf("1E5  === dp\n");
	//
	FOR(0, i, mdl->insts) free(grad_1e5[i]);
	cudafree<uint>(ts__d);
};