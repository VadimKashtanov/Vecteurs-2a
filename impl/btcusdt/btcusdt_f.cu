#include "hip/hip_runtime.h"
#include "btcusdt.cuh"

#include "../impl_template/tmpl_etc.cu"

static __global__ void k__f_btcusdt(
	float * somme_score,
	float * y, float * p1p0,
	uint * ts__d,
	uint Y,
	uint T)
{
	uint t = threadIdx.x + blockIdx.x * blockDim.x;
	//
	if (t < GRAND_T) {
		//float usd = Usdt;	//	100$
		//float max = Usdt;
		//
		float s=0;
		//
		FOR(0, i, DECODEUR) {
			uint mega_t = ENCODEUR + i;
			//
			uint ty        = t_MODE(t, mega_t);
			uint t_btcusdt = ts__d[t] + mega_t;
			assert(t_btcusdt < T);
			//
			//
			float _y = y[ty*Y + 0];
			assert(_y >= -1 && _y <= +1);
			//
			float _p1p0 = p1p0[t_btcusdt*1 + mega_t];
			//
			//s += powf(_y - sng(_p1p0), 2)/2 * (powf(1+fabs(_p1p0)*100, 3.0)-1);
			s += S(_y, _p1p0);
			//
			//
			//usd = usd + usd * _y * _p1p0  * Levier;
			//max = max + max * fabs(_p1p0) * Levier;
			//
			//float _S = S(A, _y, _p1p0);
			//assert(_S >= 0);
		}
		//
		atomicAdd(&somme_score[0], s);//powf(usd/max - max/max, 2)/2);
	}
};

float f_btcusdt(BTCUSDT_t * btcusdt, float * y__d, uint * ts__d) {
	uint Y = btcusdt->Y;
	//
	//
	float * somme__d = cudalloc<float>(1);
	k__f_btcusdt<<<dim3(KERD(GRAND_T, 16)), dim3(16)>>>(
		somme__d,
		y__d, btcusdt->sorties__d,
		ts__d,
		Y,
		btcusdt->T
	);
	ATTENDRE_CUDA();
	//
	//
	float * somme = gpu_vers_cpu<float>(somme__d, 1);
	//
	float score = somme[0] / ((float)(GRAND_T * DECODEUR));
	//
	//
	cudafree<float>(somme__d   );
	    free       (somme      );
	//
	return score;
};