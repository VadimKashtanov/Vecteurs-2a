#include "hip/hip_runtime.h"
#include "btcusdt.cuh"

#include "../impl_template/tmpl_etc.cu"

static __global__ void k__df_btcusdt(
	float * y, float * p1p0, float * dy,
	uint * ts__d,
	uint Y,
	uint T)
{
	uint t = threadIdx.x + blockIdx.x * blockDim.x;
	//
	if (t < GRAND_T) {
		//float usd = Usdt;	//	100$
		//float max = Usdt;
		//
		FOR(0, i, DECODEUR) {
			uint mega_t = ENCODEUR + i;
			//
			uint ty        = t_MODE(t, mega_t);
			uint t_btcusdt = ts__d[t] + mega_t;
			assert(t_btcusdt < T);
			//
			//
			float _y = y[ty*Y + 0];
			assert(_y >= -1 && _y <= +1);
			//
			float _p1p0 = p1p0[t_btcusdt*1 + mega_t];
			//
			//dy[ty*Y + 0] = (_y - sng(_p1p0))  * (powf(1+fabs(_p1p0)*100, 3.0)-1)  / ((float)(GRAND_T * DECODEUR));
			//
			float _ds = dS(_y, _p1p0);
			float diviseur = ((float)(GRAND_T * DECODEUR));
			dy[ty*Y + 0] = _ds / diviseur;
			//
			//
			//usd = usd + usd * _y * _p1p0  * Levier;
			//max = max + max * fabs(_p1p0) * Levier;
		}
		//
		/*float s  = powf(usd/max - max/max, 2)/2;
		float ds = (usd/max - max/max) * 1/max;
		//
		float dusdt = ds;
		//
		RETRO_FOR(0, i, DECODEUR) {
			uint mega_t = ENCODEUR + i;
			//
			uint ty        = t_MODE(t, mega_t);
			uint t_btcusdt = ts__d[t] + mega_t;
			assert(t_btcusdt < T);
			//
			//
			float _y = y[ty*Y + 0];
			assert(_y >= -1 && _y <= +1);
			//
			float _p1p0 = p1p0[t_btcusdt*1 + mega_t];
			//
			//
			//usd = usd*(1 + _y * _p1p0  * L);
			float usdt_avant = usd / (1 + _y * _p1p0 * Levier);
			dy[ty*Y + 0] = dusdt * usd * _p1p0 * Levier;
			//
			dusdt = dusdt * (1 + _y*_p1p0*Levier);
			usd = usdt_avant; 
			//max = max + max * fabs(_p1p0) * L;
		}*/
	}
};

void df_btcusdt(BTCUSDT_t * btcusdt, float * y__d, float * dy__d, uint * ts__d) {
	uint Y = btcusdt->Y;
	//
	//
	k__df_btcusdt<<<dim3(KERD(GRAND_T, 16)), dim3(16)>>>(
		y__d, btcusdt->sorties__d, dy__d,
		ts__d,
		Y,
		btcusdt->T
	);
	ATTENDRE_CUDA();
};